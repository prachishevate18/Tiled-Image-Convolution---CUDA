#include "hip/hip_runtime.h"
﻿#include <wb.h>

#define MASK_WIDTH 5
#define O_TILE_WIDTH 16
#define BLOCK_WIDTH O_TILE_WIDTH + MASK_WIDTH -1
#define MASK_RADIUS MASK_WIDTH/2
#define clamp(x) (min(max((x), 0.0), 1.0))

//@@ INSERT CODE HERE 
//implement the tiled 2D convolution kernel with adjustments for channels
//use shared memory to reduce the number of global accesses, handle the boundary conditions when loading input list elements into the shared memory
//clamp your output values

__global__ void convolution_2D_kernel(float *P, float* __restrict__ M, float *N, int height, int width, int channels)
{
	__shared__ float Ns[BLOCK_WIDTH][BLOCK_WIDTH];  //Using shared memory
	int k, ty = threadIdx.y, tx = threadIdx.x;
	int row_o = blockIdx.y*O_TILE_WIDTH + ty;
	int col_o = blockIdx.x*O_TILE_WIDTH + tx;
	int row_i = row_o - 2;
	int col_i = col_o - 2;


	for (k = 0; k < channels; k++)
	{
		float accum = 0.0f;

		//Boundary condition when loading input list elements into the shared memory
		if ((row_i >= 0) && (row_i < height) && (col_i >= 0) && (col_i < width))
		{
			Ns[ty][tx] = N[(row_i * width + col_i)* channels + k];
		}
		else
		{
			Ns[ty][tx] = 0.0f;
		}
		__syncthreads();

		if (ty < O_TILE_WIDTH && tx < O_TILE_WIDTH)
		{
			for (int i = 0; i <MASK_WIDTH; i++)
			{
				for (int j = 0; j < MASK_WIDTH; j++)
				{
					accum += M[i*MASK_WIDTH + j] * Ns[i + ty][j + tx];
				}
			}
			//@@ pixels are in the range of 0 to 1
			if (row_o < height && col_o < width)
			{
				P[(row_o * width + col_o)*channels + k] = clamp(accum); //clamping the output values
			}
		}
		__syncthreads();
	}
}

int main(int argc, char *argv[]) {
	wbArg_t arg;
	int maskRows;
	int maskColumns;
	int imageChannels;
	int imageWidth;
	int imageHeight;
	char *inputImageFile;
	char *inputMaskFile;
	wbImage_t inputImage;
	wbImage_t outputImage;
	float *hostInputImageData;
	float *hostOutputImageData;
	float *hostMaskData;
	float *deviceInputImageData;
	float *deviceOutputImageData;
	float *deviceMaskData;

	arg = wbArg_read(argc, argv); /* parse the input arguments */

	inputImageFile = wbArg_getInputFile(arg, 0);
	inputMaskFile = wbArg_getInputFile(arg, 1);

	inputImage = wbImport(inputImageFile);
	hostMaskData = (float *)wbImport(inputMaskFile, &maskRows, &maskColumns);

	assert(maskRows == MASK_WIDTH);    /* mask height is fixed to 5 */
	assert(maskColumns == MASK_WIDTH); /* mask width is fixed to 5 */

	imageWidth = wbImage_getWidth(inputImage);
	imageHeight = wbImage_getHeight(inputImage);
	imageChannels = wbImage_getChannels(inputImage);

	outputImage = wbImage_new(imageWidth, imageHeight, imageChannels);

	hostInputImageData = wbImage_getData(inputImage);
	hostOutputImageData = wbImage_getData(outputImage);

	wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

	wbTime_start(GPU, "Doing GPU memory allocation");
	//@@ INSERT CODE HERE
	//allocate device memory
	hipMalloc((void **)&deviceInputImageData, imageWidth * imageHeight * imageChannels * sizeof(float));
	hipMalloc((void **)&deviceOutputImageData, imageWidth * imageHeight * imageChannels * sizeof(float));
	hipMalloc((void **)&deviceMaskData, maskRows * maskColumns * sizeof(float));
	wbTime_stop(GPU, "Doing GPU memory allocation");

	wbTime_start(Copy, "Copying data to the GPU");
	//@@ INSERT CODE HERE
	//copy host memory to device
	hipMemcpy(deviceInputImageData, hostInputImageData, imageWidth * imageHeight * imageChannels * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(deviceMaskData, hostMaskData, maskRows * maskColumns * sizeof(float), hipMemcpyHostToDevice);

	wbTime_stop(Copy, "Copying data to the GPU");

	wbTime_start(Compute, "Doing the computation on the GPU");
	//@@ INSERT CODE HERE
	//initialize thread block and kernel grid dimensions
	//invoke CUDA kernel	
	dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH);
	dim3 dimGrid((wbImage_getWidth(inputImage) - 1) / O_TILE_WIDTH + 1, (wbImage_getHeight(inputImage) - 1) / O_TILE_WIDTH + 1, 1);
	convolution_2D_kernel<<<dimGrid, dimBlock>>>(deviceOutputImageData, deviceMaskData, deviceInputImageData, imageHeight, imageWidth, imageChannels);
	wbTime_stop(Compute, "Doing the computation on the GPU");

	wbTime_start(Copy, "Copying data from the GPU");
	//@@ INSERT CODE HERE
	//copy results from device to host	
	hipMemcpy(hostOutputImageData, deviceOutputImageData, imageWidth * imageHeight * imageChannels * sizeof(float), hipMemcpyDeviceToHost);

	wbTime_stop(Copy, "Copying data from the GPU");

	wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

	wbSolution(arg, outputImage);

	//@@ INSERT CODE HERE
	//deallocate device memory
	hipFree(deviceInputImageData);
	hipFree(deviceOutputImageData);
	hipFree(deviceMaskData);


	free(hostMaskData);
	wbImage_delete(outputImage);
	wbImage_delete(inputImage);

	return 0;
}